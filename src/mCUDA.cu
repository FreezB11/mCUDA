#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mCUDA.h"

__global__ void addKernel(float* a, float* b, float* result, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        result[idx] = a[idx] + b[idx];
    }
}

void launchAddKernel(float* a, float* b, float* result, int n) {
    float *d_a, *d_b, *d_result;

    size_t size = n * sizeof(float);
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_result, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    addKernel<<<blocks, threads>>>(d_a, d_b, d_result, n);

    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
