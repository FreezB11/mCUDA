#include <hip/hip_runtime.h>
#include <cstring>

__global__ void matMulKernel(const float* A, const float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

// GPU wrapper
void gpuMatrixMul(const float* A, const float* B, float* C, int n) {
    size_t bytes = n * n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((n + 15) / 16, (n + 15) / 16);

    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
