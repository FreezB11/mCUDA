// main.cu
#include <iostream>
#include "learn/mCUDA.cuh"

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Call vector addition
    vectorAdd(d_A, d_B, d_C, N);

    // Copy result back
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print some results
    for (int i = 0; i < 5; ++i) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;

    return 0;
}
